/**
 * @author Luong Huu Phuc
 * @file neces_lib.cpp
 */
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <>
#include "neces_lib.cuh"

hipEvent_t start_time = nullptr;
hipEvent_t end_time = nullptr;
float elasped_time = 0.0f;
int count = 0;
std::chrono::high_resolution_clock::time_point tStart;

__host__ void startTimer(void){
  hipEventCreate(&start_time);
  hipEventCreate(&end_time);
  hipEventRecord(start_time, 0);
}

__host__ double elaspedTimer(void){
  hipEventRecord(end_time, 0);
  hipEventSynchronize(end_time); //Dam bao kernel da chay xong
  
  hipEventElapsedTime(&elasped_time, start_time, end_time); //thoi gian tinh bang milliseconds
  hipEventDestroy(start_time);
  hipEventDestroy(end_time);

  return elasped_time;
}

__host__ void swapPointers(long **a, long **b){
  long *temp = *a; 
  *a = *b; 
  *b = temp;
}

void printArray(long *arr, int n){
  for(int i = 0; i < n; i++){
     std::cout << arr[i] << " ";
     count++;
     if(count == 20){
      std::cout << std::endl;
      count = 0;
     }
  }
}

int tm(void){
  static bool initialized = false;
  if(!initialized){
    tStart = std::chrono::high_resolution_clock::now();
    initialized = true;
    return 0;
  }

  auto tEnd = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(tEnd - tStart).count();
  tStart = tEnd;

  return static_cast<int>(duration); //Tra ve microseconds
}