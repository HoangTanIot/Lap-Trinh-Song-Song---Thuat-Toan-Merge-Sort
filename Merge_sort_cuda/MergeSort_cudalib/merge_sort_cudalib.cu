#include "hip/hip_runtime.h"
/**
 * @file merge_sort_cudalib.cu
 * @date 2025/04/05
 * @author Luong Huu Phuc 
 * @copyright JoeyOhman
 * \anchor https://github.com/JoeyOhman/GPUMergeSort
 */
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>
#include "neces_lib.cuh"
#include "merge_sort_cudalib.cuh"
#include "Cuda_check_err.cu"

//Ham kiem tra xem thu tu phan tu da chuan chua
__host__ bool isSorted(long *arr, long n){
  for(int i = 1; i < n; i++){
    // if(arr[i] != i){
    //   return 0;
    // }
    if(arr[i - 1] > arr[i]){
      return 0;
    }
  }
  return 1;
}

/************************** VERSION 1 ***************************/

__device__ int binarySearch(long *arr, int val, int left, int right){
  if(right <= left){ //Khi index right == index left (con 1 phan tu)
    return (val > arr[left] ? (left + 1) : left); //Neu gia tri can tim (val) lon hon gia tri left
  }
  int mid = (left + right) / 2;
  if(val > arr[mid]){ //Neu gia tri can tim lon hon gia tri o giua 
    return binarySearch(arr, val, mid + 1, right); //Bo het mang ben trai (be hon val), de quy tiep mang ben phai
  }
  return binarySearch(arr, val, left, mid); //Khong thi bo het mang ben phai (lon hon val), de quy tiep mang ben trai
}

__device__ int getIndex(long *subAux, int ownIndex, int nLeft, int nTot){
  int scanIndex;
  int upperBound;
  bool partOfFistArr = ownIndex < nLeft; //Xem xem phan tu hien tai thuoc mang ben nao (xet index cua subAux)

  if(partOfFistArr){ //Dung => Phan tu thuoc mang trai => tim xem bao nhieu phan tu ben phai nho hon no
    scanIndex = nLeft;
    upperBound = nTot;
  }else{
    scanIndex = 0;
    upperBound = nLeft;
  }

  scanIndex = binarySearch(subAux, subAux[ownIndex], scanIndex, upperBound - 1);
  return ownIndex + scanIndex - nLeft;
} 

__global__ void mergeKernel(long *arr, long *aux, int left, int mid, int right){
  /**
   * @brief Tinh so thread trong gird cua CUDA, moi thread phu trach 1 phan tu tu aux[left..]
   * @param blockIdx Chi so block moi grid (0 -> gridDim.x - 1)
   * @param blockDim So thread moi block 
   * @param threadIdx Chi so thread hien tai trong block do (0 -> blockDim.x - 1)
   */
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int nLeft = mid - left + 1;
  int nRight = right - mid;
  int nTot = nLeft + nRight;

  //Neu thread co chi so vuot ngoai tong so phan tu thi dung lai - khong lam gi ca
  if(idx >= nTot){
    return;
  }

  //Xac dinh phan tu aux[left + idx] dang thuoc mang con trai hay phai
  //Sau do se thuc hien tim kiem nhi phan trong mang con lai de dem xem co bao nhieu phan tu nho hon no
  //Nham xac dinh vi tri chinh xac sau khi merge
  int arrIndex = getIndex(&aux[left], idx, nLeft, nTot);
  arr[left + arrIndex] = aux[left + idx]; //Ghi phan tu vao dung vi tri trong mang arr

  //Loi dong nay khi build vi std::cout la ham cua CPU, khong duoc phep dung trong __global__ hay __device__
  // std::cout << "Index " << idx << " assigns " << aux[left + idx] << " to " << left + arrIndex << std::endl;
}

__host__ __device__ void merge(long *arr, long *aux, int left, int mid, int right){
  int i = 0;
  int j = 0;
  int mergeIndex = left;
  int nLeft = mid - left + 1; //So phan tu mang ben trai
  int nRight = right - mid; //So phan tu mang ben phai

  while(i < nLeft && j < nRight){
    if(aux[left + i] < aux[mid + 1 + j]){ //Neu phan tu nao mang ben trai < be hon ben phai
      arr[mergeIndex] = aux[left + i]; //Day vao mang arr 
      i++;
    }else{
      arr[mergeIndex] = aux[mid + 1 + j]; //Khong thi day phan tu ben phai vao mang
      j++;
    }
    mergeIndex++;
  }

  while(i < nLeft){
    arr[mergeIndex] = aux[left + i];
    i++;
    mergeIndex++;
  }

  while(j < nRight){
    arr[mergeIndex] = aux[mid + 1 + j];
    j++;
    mergeIndex++;
  }
}

__global__ void mergeSort(long *arr, long *aux, int currentSize, int n, int width){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 

  int left = idx * width; //Chi so bat dau cua doan dang xet

  if(left >= n - currentSize || left < 0){
    return;
  }

  int mid = left + currentSize - 1; //Ket thuc cua doan thu nhat 
  int right = min_local(left + width - 1, n - 1);

  int nTot = right - left + 1; //So threads duoc sinh ra

  if(nTot > 16384){ //Neu phan tu lon hon 16384 (nguong de co the song song kernel)
    int numThreadsPerBlock = 1024; //1024 thread moi block
    int numBlocks = (nTot + numThreadsPerBlock - 1) / numThreadsPerBlock; //So blocks duoc sinh ra theo so phan tu mang

    mergeKernel <<< numBlocks, numThreadsPerBlock >>>(arr, aux, left, mid, right);
    cudaCheckErrorDev();
  }else{ //Neu phan tu nho hon nguong quy dinh (de tranh overhead)
    merge(arr, aux, left, mid, right);
  }
}

void mergeSortGPU(long *arr, int n){
  //Hai mang trong device (GPU)
  long *deviceArr;
  long *auxArr;

  //Uu tien su dung cache L1 hon shared memory -> Toi uu hieu suat cho kernel nho
  cudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  cudaSafeCall(hipMalloc((void**)&deviceArr, n * sizeof(int)));
  cudaSafeCall(hipMalloc((void**)&auxArr, n * sizeof(int)));
  cudaSafeCall(hipMemcpy(deviceArr, arr, n * sizeof(int), hipMemcpyHostToDevice)); //Sao chep du lieu tu mang arr(host - CPU) vao deviceArr(GPU)

  //Duyet qua cac kich thuoc doan con: 1, 2, 4, 8,...n
  //Moi lan lap se merge cac doan co cung kich thuoc
  for(int currentSize = 1; currentSize < n; currentSize *= 2){
    //Tinh toan tham so kernel
    int width = currentSize * 2; 
    int numSorts = (n + width - 1) / width; //So luong sorting thread sinh ra (so merge can thuc hien)
    int numThreadsPerBlock = 32;
    int numBlocks = (numSorts + numThreadsPerBlock - 1) / numThreadsPerBlock;

    cudaSafeCall(hipMemcpy(auxArr, deviceArr, n * sizeof(int), hipMemcpyDeviceToDevice));
    mergeSort <<< numBlocks, numThreadsPerBlock >>> (deviceArr, auxArr, currentSize, n, width);
    hipDeviceSynchronize(); //__host__ function
    cudaCheckError();
  }

  //Sau khi sap xep xong thi tra ve arr tren CPU 
  cudaSafeCall(hipMemcpy(arr, deviceArr, n * sizeof(int), hipMemcpyDeviceToHost));

  cudaSafeCall(hipFree(deviceArr));
  cudaSafeCall(hipFree(auxArr));
}

/************************** VERSION 2 ***************************/

__device__ unsigned int getIndex_kernel(dim3 *threads, dim3 *blocks){
  int x;
  return (threadIdx.x + 
          threadIdx.y * (x = threads->x) + 
          threadIdx.z * (x *= threads->y) + 
          blockIdx.x * (x *= threads->z) + 
          blockIdx.y * (x *= blocks->z) +
          blockIdx.z * (x *= blocks->y));
}

__device__ void gpu_bottomUpMerge_ver2(long *arr, long *aux, long left, long mid, long right){
  long i = left;
  long j =  mid;
  for(long k = left; k < right; k++){
    if(i < mid && (j >= right || arr[i] < arr[j])){
      aux[k] = arr[i];
      i++;
    }else{
      aux[k] = arr[j];
      j++;
    }
  }
}

__global__ void gpu_mergeSort_ver2(long *arr, long *aux, long n, long width, long slices, dim3 *threads, dim3 *blocks){
  unsigned int idx = getIndex_kernel(threads, blocks);
  long left = width * idx * slices, mid, right;

  for(long slice = 0; slice < slices; slice++){
    if(left >= n){
      break;
    }
    mid = min_local(left + (width >> 1), n);
    right = min_local(left + width, n);
    gpu_bottomUpMerge_ver2(arr, aux, left, mid, right);
    left += width;
  }
}

void mergeSortGPU_ver2(long *arr, long n, dim3 threadsPerBlock, dim3 blocksPerGrid){
  long *deviceArr;
  long *auxArr;
  dim3 *deviceThreads;
  dim3 *deviceBlocks;

  // tm();

  cudaSafeCall(hipMalloc((void**)&deviceArr, n * sizeof(long)));
  cudaSafeCall(hipMalloc((void**)&auxArr, n * sizeof(long)));
  cudaSafeCall(hipMemcpy(deviceArr, arr, n * sizeof(long), hipMemcpyHostToDevice));

  cudaSafeCall(hipMalloc((void**)&deviceThreads, sizeof(dim3)));
  cudaSafeCall(hipMalloc((void**)&deviceBlocks, sizeof(dim3)));
  cudaSafeCall(hipMemcpy(deviceThreads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(deviceBlocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice));

  long *A = deviceArr;
  long *B = auxArr;

  long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z * 
                 blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

  for(int width = 2; width < (n << 1); width <<= 1){
    long slices = n / ((nThreads) * width) + 1;

    gpu_mergeSort_ver2 <<< blocksPerGrid, threadsPerBlock >>>(A, B, n, width, slices, deviceThreads, deviceBlocks);

    A = A == deviceArr ? auxArr : deviceArr;
    B = B == deviceArr ? auxArr : deviceArr;
  }

  // tm();
  cudaSafeCall(hipMemcpy(arr, A, n * sizeof(int), hipMemcpyDeviceToHost));

  cudaSafeCall(hipFree(deviceArr));
  cudaSafeCall(hipFree(auxArr));
}