#include "hip/hip_runtime.h"
/**
 * @file merge_sort_cudalib.cu
 * @date 2025/04/05
 * @author Luong Huu Phuc 
 * @copyright JoeyOhman
 * \anchor https://github.com/JoeyOhman/GPUMergeSort
 */
#include <iostream>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <>
#include "neces_lib.cuh"
#include "merge_sort_cudalib.cuh"
#include "Cuda_check_err.cu"

//Ham kiem tra xem thu tu phan tu da chuan chua
__host__ bool isSorted(long *arr, long n){
  for(int i = 1; i < n; i++){
    // if(arr[i] != i){
    //   return 0;
    // }
    if(arr[i - 1] > arr[i]){
      return 0;
    }
  }
  return 1;
}

/************************** VERSION 1 ***************************/

__device__ int binarySearch(long *arr, int val, int left, int right){
  if(right <= left){ //Khi index right == index left (con 1 phan tu)
    return (val > arr[left] ? (left + 1) : left); //Neu gia tri can tim (val) lon hon gia tri left
  }
  int mid = (left + right) / 2;
  if(val > arr[mid]){ //Neu gia tri can tim lon hon gia tri o giua 
    return binarySearch(arr, val, mid + 1, right); //Bo het mang ben trai (be hon val), de quy tiep mang ben phai
  }
  return binarySearch(arr, val, left, mid); //Khong thi bo het mang ben phai (lon hon val), de quy tiep mang ben trai
}

__device__ int getIndex(long *subAux, int ownIndex, int nLeft, int nTot){
  int scanIndex; //Bien luu gioi han duoi cho binary search
  int upperBound; //Bien luu gioi han tren cho binary search 
  bool isLeft = ownIndex < nLeft; //Xem xem phan tu hien tai thuoc mang ben nao (xet index cua subAux)

  if(isLeft){ //Dung => Phan tu thuoc mang trai => tim xem bao nhieu phan tu ben PHAI nho hon no
    scanIndex = nLeft; //la chi so dau tien cua mang phai 
    upperBound = nTot; //Gioi han tren (diem cuoi cua mang phai)
  }else{ //Neu phan tu thuoc mang phai => tim xem bao nhieu phan tu ben TRAI nho hon no
    scanIndex = 0; //la chi so dau tien cua mang trai
    upperBound = nLeft; //sau cuoi mang trai
  }

  //Tim kiem bang Binary Search cho 2 nua TRAI hoac PHAI do 
  scanIndex = binarySearch(subAux, subAux[ownIndex], scanIndex, upperBound - 1); //Tra ve so luong phan tu nho hon tu phia con lai
  return ownIndex + scanIndex - nLeft; //Tru di nLeft vi mang subAux dang duoc danh chi so tu [0]
} 

__global__ void mergeKernel(long *arr, long *aux, int left, int mid, int right){
  /**
   * @brief Tinh so thread trong gird cua CUDA, moi thread phu trach 1 phan tu tu aux[left..]
   * @param blockIdx Chi so block moi grid (0 -> gridDim.x - 1)
   * @param blockDim So thread moi block 
   * @param threadIdx Chi so thread hien tai trong block do (0 -> blockDim.x - 1)
   */
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  int nLeft = mid - left + 1; //Xac dinh vi tri ket thuc cua mang ben trai (= So luong phan tu mang trai)
  int nRight = right - mid; //Xac dinh vi tri bat dau cua mang ben phai (= So luong phan tu mang phai)
  int nTot = nLeft + nRight; //Tong so phan tu cua ca 2 mang 

  //Neu thread co chi so vuot ngoai tong so phan tu thi dung lai - khong lam gi ca
  if(idx >= nTot){
    return;
  }

  //Xac dinh phan tu aux[left + idx] dang thuoc mang con trai hay phai
  //Sau do se thuc hien tim kiem nhi phan trong mang con lai de dem xem co bao nhieu phan tu nho hon no
  //Nham xac dinh vi tri chinh xac sau khi merge roi push vao trong mang arr cuoi cung
  int arrIndex = getIndex(&aux[left], idx, nLeft, nTot);
  arr[left + arrIndex] = aux[left + idx]; //Ghi phan tu vao dung vi tri trong mang arr
}

__host__ __device__ void merge(long *arr, long *aux, int left, int mid, int right){
  int i = 0;
  int j = 0;
  int mergeIndex = left;
  int nLeft = mid - left + 1; //So phan tu mang ben trai
  int nRight = right - mid; //So phan tu mang ben phai

  while(i < nLeft && j < nRight){
    if(aux[left + i] < aux[mid + 1 + j]){ //Neu phan tu nao mang ben trai < be hon ben phai
      arr[mergeIndex] = aux[left + i]; //Day vao mang arr 
      i++;
    }else{
      arr[mergeIndex] = aux[mid + 1 + j]; //Khong thi day phan tu ben phai vao mang
      j++;
    }
    mergeIndex++;
  }

  while(i < nLeft){
    arr[mergeIndex] = aux[left + i];
    i++;
    mergeIndex++;
  }

  while(j < nRight){
    arr[mergeIndex] = aux[mid + 1 + j];
    j++;
    mergeIndex++;
  }
}

__global__ void mergeSort_coordinate(long *arr, long *aux, int currentSize, int n, int afterSize){
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 

  int left = idx * afterSize; //Chi so bat dau cua doan dang xet
  /**
   * Thread thu idx se thuc hien gop 2 mang con bat dau tu phan tu `left`
   * Moi mang con dai currentSize, nen thread nay se merge doan tu `left` den `left + afterSize - 1`
   * Gia su: currentSize = 2 -> afterSize = currentSize * 2 = 4 (sau khi gop mang 2 phan tu thi se duoc mang co do dai 4 phan tu), idx = 3
   * => left = idx * afterSize = 3 * 4 = 12;
   * => mid = left + currentSize - 1 = 12 + 2 - 1 = 13
   * => right = left + afterSize - 1 = 12 + 4 - 1 = 15
   * -> Thread SO 3 se merge 2 doan con:
   *  * Mang con trai: arr[12], arr[13]
   *  * Mang con phai: arr[14], arr[15]
   */

  //Neu left nam ngoai pham vi mang
  if(left >= n - currentSize || left < 0){
    return;
  }

  int mid = left + currentSize - 1; //Ket thuc cua mang con ben trai
  int right = min_local(left + afterSize - 1, n - 1); //Ket thuc cua mang con ben phai

  int nTot = right - left + 1; //So threads duoc sinh ra (Tong so phan tu trong doan [left, right])

  if(nTot > MERGE_PARALLEL_THRESHOLD){ //Neu so luong phan tu can merge lon hon nguong de co the song song kernel ma khong gay overhead
    int numThreadsPerBlock = 1024; //1024 thread moi block
    int numBlocks = (nTot + numThreadsPerBlock - 1) / numThreadsPerBlock; //So blocks duoc sinh ra theo so phan tu mang

    mergeKernel <<< numBlocks, numThreadsPerBlock >>>(arr, aux, left, mid, right);
    cudaCheckErrorDev(); //Check loi sau moi lan goi kernel
  }else{ //Neu phan tu nho hon nguong quy dinh (de tranh overhead)
    merge(arr, aux, left, mid, right);
  }
}

void mergeSortGPU(long *arr, int n){
  //Hai mang trong device (GPU)
  long *deviceArr; //Mang chua phan tu sau khi da sap xep xong (cuoi cung)
  long *auxArr; //Mang trung gian luu ket qua cua tung buoc hop nhat

  //Uu tien su dung cache L1 hon shared memory -> Toi uu hieu suat cho kernel nho
  cudaSafeCall(hipDeviceSetCacheConfig(hipFuncCachePreferL1));
  cudaSafeCall(hipMalloc((void**)&deviceArr, n * sizeof(long)));
  cudaSafeCall(hipMalloc((void**)&auxArr, n * sizeof(long)));
  cudaSafeCall(hipMemcpy(deviceArr, arr, n * sizeof(long), hipMemcpyHostToDevice)); //Sao chep du lieu tu mang arr(host - CPU) vao deviceArr(GPU)

  //currentSize: kich thuoc hien tai cua mang con dang merge
  //Duyet qua cac mang con co kich thuoc currentSize = 1, 2, 4, 8,...n
  //Moi lan lap se merge cac doan co do dai currentSize, sau moi vong lap so phan tu trong mang se tang len 2 lan
  for(int currentSize = 1; currentSize < n; currentSize *= 2){
    //Tinh toan tham so kernel
    int afterSize = currentSize * 2; //Bien luu tong do dai cua 2 mang con sau khi duoc merge (2 -> 4 -> 8,...)
    int numSorts = (n + afterSize - 1) / afterSize; //So luong sorting thread sinh ra (so merge can thuc hien)
    int numThreadsPerBlock = 32; 
    int numBlocks = (numSorts + numThreadsPerBlock - 1) / numThreadsPerBlock;

    cudaSafeCall(hipMemcpy(auxArr, deviceArr, n * sizeof(long), hipMemcpyDeviceToDevice)); //Truoc khi hop nhat copy du lieu tu deviceArr sang auxArr
    //Trong moi vong lap currentSize, no goi kernel mergeSort de xu ly nhieu doan nho song song 
    //Trong kernel mergeSort, neu doan mang du lon thi ta tiep tuc goi 1 kernel con mergeKernel() de song song hoa qua trinh merge tung phan tu trong doan do
    mergeSort_coordinate <<< numBlocks, numThreadsPerBlock >>> (deviceArr, auxArr, currentSize, n, afterSize);
    hipDeviceSynchronize(); //__host__ function
    cudaCheckError();
  }

  //Sau khi sap xep xong thi tra ve arr tren CPU 
  cudaSafeCall(hipMemcpy(arr, deviceArr, n * sizeof(long), hipMemcpyDeviceToHost));

  cudaSafeCall(hipFree(deviceArr));
  cudaSafeCall(hipFree(auxArr));
}

/************************** VERSION 2 ***************************/

__device__ unsigned int getIndex_kernel(dim3 *threads, dim3 *blocks){
  int x;
  return (threadIdx.x + 
          threadIdx.y * (x = threads->x) + 
          threadIdx.z * (x *= threads->y) + 
          blockIdx.x * (x *= threads->z) + 
          blockIdx.y * (x *= blocks->z) +
          blockIdx.z * (x *= blocks->y));
}

__device__ void gpu_bottomUpMerge_ver2(long *arr, long *aux, long left, long mid, long right){
  long i = left;
  long j =  mid;
  for(long k = left; k < right; k++){
    if(i < mid && (j >= right || arr[i] < arr[j])){
      aux[k] = arr[i];
      i++;
    }else{
      aux[k] = arr[j];
      j++;
    }
  }
}

__global__ void gpu_mergeSort_ver2(long *arr, long *aux, long n, long afterSize, long slices, dim3 *threads, dim3 *blocks){
  unsigned int idx = getIndex_kernel(threads, blocks);
  long left = afterSize * idx * slices, mid, right;

  for(long slice = 0; slice < slices; slice++){
    if(left >= n){
      break;
    }
    mid = min_local(left + (afterSize >> 1), n);
    right = min_local(left + afterSize, n);
    gpu_bottomUpMerge_ver2(arr, aux, left, mid, right);
    left += afterSize;
  }
}

void mergeSortGPU_ver2(long *arr, long n, dim3 threadsPerBlock, dim3 blocksPerGrid){
  long *deviceArr;
  long *auxArr;
  dim3 *deviceThreads;
  dim3 *deviceBlocks;

  // tm();

  cudaSafeCall(hipMalloc((void**)&deviceArr, n * sizeof(long)));
  cudaSafeCall(hipMalloc((void**)&auxArr, n * sizeof(long)));
  cudaSafeCall(hipMemcpy(deviceArr, arr, n * sizeof(long), hipMemcpyHostToDevice));

  cudaSafeCall(hipMalloc((void**)&deviceThreads, sizeof(dim3)));
  cudaSafeCall(hipMalloc((void**)&deviceBlocks, sizeof(dim3)));
  cudaSafeCall(hipMemcpy(deviceThreads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice));
  cudaSafeCall(hipMemcpy(deviceBlocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice));

  long *A = deviceArr;
  long *B = auxArr;

  long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z * 
                 blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

  for(int afterSize = 2; afterSize < (n << 1); afterSize <<= 1){
    long slices = n / ((nThreads) * afterSize) + 1;

    gpu_mergeSort_ver2 <<< blocksPerGrid, threadsPerBlock >>>(A, B, n, afterSize, slices, deviceThreads, deviceBlocks);

    A = A == deviceArr ? auxArr : deviceArr;
    B = B == deviceArr ? auxArr : deviceArr;
  }

  // tm();
  cudaSafeCall(hipMemcpy(arr, A, n * sizeof(int), hipMemcpyDeviceToHost));

  cudaSafeCall(hipFree(deviceArr));
  cudaSafeCall(hipFree(auxArr));
}