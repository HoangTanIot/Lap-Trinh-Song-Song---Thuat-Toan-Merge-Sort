#include "hip/hip_runtime.h"
/**
 * @brief Bottom-Up merge sort on GPU 
 * @author Luong Huu Phuc
 * @copyright Joey Ohman
 */
#include <iostream>
#include <fstream>
#include <filesystem>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <>
#include "pseudo_num_gen.h"
#include "neces_lib.cuh"
#include "merge_sort_cudalib.cuh"

//Version nay chay duoc 10 trieu phan tu -> 100tr bi crash
double benchmarkGPU(long *arr, int n){
  startTimer();
  mergeSortGPU(arr, n);
  return (elaspedTimer() / 1000.0f); //Tra ve giay
}

//Version nay chi chay duoc 1 trieu phan tu la max 
double benchmarkGPU_ver2(long *arr, long n, dim3 threads, dim3 blocks){
  startTimer();
  mergeSortGPU_ver2(arr, n, threads, blocks);
  return (elaspedTimer() / 1000.0f);
}

int main(void){
  long num;
  const int numSorts = 15;
  double num_gen_time, total_time = 0.0f;

  // dim3 threadsPerBlock, blocksPerGrid;

  // threadsPerBlock.x = 32;
  // threadsPerBlock.y = 1;
  // threadsPerBlock.z = 1;

  // blocksPerGrid.x = 8;
  // blocksPerGrid.y = 1;
  // blocksPerGrid.z = 1;

  std::cout << "Nhap kich thuoc mang: ";
  std::cin >> num;
  std::cin.ignore();

  long *arr = new long[num];
  if(arr == NULL){
    std::cerr << "Mang cap phat khong thanh cong !" << std::endl;
    return 1;
  }

  //Tao folder Time_test neu chua ton tai
  std::filesystem::create_directories("D:/C-C++_project/Project_2024-2/Merge_sort_cuda/main/Time_test");

  std::ofstream outFile("D:/C-C++_project/Project_2024-2/Merge_sort_cuda/main/Time_test/result.csv");
  if(!outFile.is_open()){
    std::cerr << "Khong the mo file de ghi !" << std::endl;
    delete[] arr;
    return 1;
  }
  outFile << "Lan,thoi_gian(s),thoi_gian_sinh_so(s)" << "\n";

  for(int i = 0; i < numSorts; i++){
    num_gen_time = pseudo_number_generate(arr, num);

    double duration = benchmarkGPU(arr, num);
    total_time += duration;

    bool check = isSorted(arr, num);
    if(!check){
      std::cerr << "Lan " << i + 1 << ": check Failed !" << std::endl;
      std::cout << "Error array: " << std::endl;
      printArray(arr, num);
      outFile << "Lan " << i + 1 << ": check Failed !" << "\n";
      delete[] arr;
      outFile.close();
      return 1;
    }

    //In ket qua ra man hinh + luu vao file (tgian thuc thi + tgian tao mang)
    std::cout << "Lan "<< i + 1 << ": " << duration << " ms, check: OK !" << std::endl;
    std::cout << "Number generate time: " << num_gen_time << " seconds" << std::endl;
    outFile << i + 1 << "," << std::fixed << std::setprecision(10) << duration << ","
                            << std::fixed << std::setprecision(10) << num_gen_time << "\n";
  }

  double average_time = total_time / numSorts;
  std::cout << "Thoi gian trung binh: " << average_time << std::endl;
  outFile << "Average," << std::fixed << std::setprecision(10) << average_time << "\n";

  delete[] arr;
  outFile.close();
  return 0;
}

/**
 * @note Loi khi chay 100 trieu phan tu la do so luong lon khien so lan goi kernel qua nhieu gay tran bo nho
 */